#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>

#define N 10

// id = blockIdx.x * blockDim.x + threadIdx.x 
// example blockId.x = 0,1,2; blokcDim.x = 5, threadIdx=0,1,2,3,4
// id 0,1,2,3,4,5,6,7,8,9,10,11,12,13,14


ada type data pada CUDA

dim3 gridDim;
dim3 blockDim;
dim3 blockIdx:
dim3 threadIdx; 


__global__ void add(int *a, int *b, int *c ){

  int tid = blockIdx.x;
  if (tid < N)
    c[tid] = a[tid] + b[tid]; 

}
  



int main(void){

  int a[N],b[N],c[N];             // varieble Host 
  int *dev_a, *dev_b, *dev_c;     // variabel GPU
  
  // allocate the memory 
  (hipMalloc)((void**)&dev_a, N * sizeof(int));
  (hipMalloc)((void**)&dev_b, N * sizeof(int));
  (hipMalloc)((void**)&dev_c, N * sizeof(int));
  
  // fill the array 'a' ad 'b' on CPU
  for (int i=0;i<N;i++){
    a[i] = -i;
    b[i] = i+1;  
      
  }
  
  // copy the array from host to device
  hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(dev_b,b,N*sizeof(int),hipMemcpyHostToDevice);
  
  add<<<N,1>>>(dev_a,dev_b,dev_c);
  
  // copy the array c from device to host 
  hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);
  
  // display the result 
  for (int i=0;i<N;i++){
    printf("%d + %d = %d \n", a[i],b[i],c[i]);
  }
  
  // free the memory allocated on GPU 
  
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  
  
  return 0;
}
