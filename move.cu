#include<hip/hip_runtime.h>
#include<stdio.h>
#include <assert.h>

int main(void)
{
printf("Getting start CUDA excercise Now \n");
float *a_h, *b_h;    // variable declartion for host (CPU)
float *a_d, *b_d;    // variabel declaration for device (GPU)
int N = 14, nBytes, i,j ;

nBytes = N * sizeof(float);
//memory allocation for CPU
a_h = (float *)malloc(nBytes);
b_h = (float *)malloc(nBytes);

// memory allocation for GPU
hipMalloc((void **) &a_d, nBytes);
hipMalloc((void **) &b_d, nBytes);

printf("Value of a_h : \n");
for (i=0; i<N; i++) 
{
  a_h[i] = 100.f + i;
  printf("%f \n", a_h[i]);    // print value of a_h
}

// start using CUDA memory copy 

hipMemcpy(a_d,a_h,nBytes,hipMemcpyHostToDevice);    // CPU to GPU
hipMemcpy(b_d,a_d,nBytes,hipMemcpyDeviceToDevice);  // GPU to GPU
hipMemcpy(b_h,b_d,nBytes,hipMemcpyDeviceToHost);


// print value of b_h

printf ("Value of b_h : \n");
for (j=0; j<N; j++) 
{
  printf("%f \n", b_h[j]);    // print value of b_h
}

// we have not to show the values of a_h and b_h 

for(int i=0; i< N; i++) assert(a_h[i]==b_h[i]);


}
